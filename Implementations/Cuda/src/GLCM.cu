#include "hip/hip_runtime.h"
#include "GLCM.h"

#include <iostream>
#include <assert.h>
#include "GLCM.h"
#include "GrayPair.h"
#include "AggregatedGrayPair.h"

using namespace std;

__host__ __device__ void checkAllocationError(GrayPair* grayPairs, AggregatedGrayPair * summed, 
    AggregatedGrayPair* subtracted, AggregatedGrayPair* xMarginal, 
    AggregatedGrayPair* yMarginal){
    if((grayPairs == NULL) || (summed == NULL) || (subtracted == NULL) ||
    (xMarginal == NULL) || (yMarginal == NULL))
        printf("ERROR: Device doesn't have enough memory");
}  


// Constructors
__device__ GLCM::GLCM(const unsigned int * pixels, const ImageData& image,
        Window& windowData, WorkArea& wa): pixels(pixels), img(image),
        windowData(windowData),  workArea(wa) ,grayPairs(wa.grayPairs),
        summedPairs(wa.summedPairs), subtractedPairs(wa.subtractedPairs),
        xMarginalPairs(wa.xMarginalPairs), yMarginalPairs(wa.yMarginalPairs)
        {
    // Compute the number of pairs that need to be processed in this GLCM
    this->numberOfPairs = getWindowRowsBorder() * getWindowColsBorder();
    if(this->windowData.symmetric)
        this->numberOfPairs *= 2;

    // Replacing dirty memory with items that represent "available memory"
    workArea.cleanup();
    // Generate elements of this GLCM
    initializeGlcmElements();}


// Set the working area to initial condition
__device__ GLCM::~GLCM(){

}

// Warning, se simmetrica lo spazio deve raddoppiare
__device__ int GLCM::getNumberOfPairs() const {
        return numberOfPairs;
}

__device__ int GLCM::getMaxGrayLevel() const {
    return img.getMaxGrayLevel();
}

// y-Side of the sub-window of interest for creating pairs
__device__ int GLCM::getWindowRowsBorder() const{
   return (windowData.side - (windowData.distance * abs(windowData.shiftRows)));
}

// x-Side of the sub-window of interest for creating pairs
__device__ int GLCM::getWindowColsBorder() const{
    return (windowData.side - (windowData.distance * abs(windowData.shiftColumns)));
}



/*
    columnOffset is a shift value used for reading the correct batch of elements
    from given linearized input pixels; for 135° the first d (distance) elements
    need to be ignored
*/
__device__ inline int GLCM::computeWindowColumnOffset()
{
    int initialColumnOffset = 0; // for 0°,45°,90°
    if((windowData.shiftRows * windowData.shiftColumns) > 0) // 135°
        initialColumnOffset = 1;
    return initialColumnOffset;
}

/*
    rowOffset is a shift value used for reading the correct batch of elements
    from given linearized input pixels according to the direction in use;
    45/90/135° must skip d (distance) "rows"
*/
__device__ inline int GLCM::computeWindowRowOffset()
{
    int initialRowOffset = 1; // for 45°,90°,135°
    if((windowData.shiftRows == 0) && (windowData.shiftColumns > 0))
        initialRowOffset = 0; // for 0°
    return initialRowOffset;
}

// addressing method for reference pixel; see documentation
__device__ inline int GLCM::getReferenceIndex(const int i, const int j,
                                   const int initialWindowRowOffset, const int initialWindowColumnOffset){
    int row = (i + windowData.imageRowsOffset) // starting point in the image
            + (initialWindowRowOffset * windowData.distance); // add direction eventual down-shift (45°, 90°, 135°)
    int col = (j + windowData.imageColumnsOffset) + // starting point in the image
            (initialWindowColumnOffset * windowData.distance); // add direction shift
    int index = ( row * img.getColumns()) + col;
    assert(index >= 0);
    return index;
}

// addressing method for neighbor pixel; see documentation
__device__ inline int GLCM::getNeighborIndex(const int i, const int j,
                                  const int initialWindowColumnOffset){
    int row = (i + windowData.imageRowsOffset); // starting point in the image
    int col = (j + windowData.imageColumnsOffset) + // starting point in the image
              (initialWindowColumnOffset * windowData.distance) +  // add 135* right-shift
              (windowData.shiftColumns * windowData.distance); // add direction shift
    int index = (row * img.getColumns()) + col;
    assert(index >= 0);
    return index;
}

/* Method that inserts a GrayPair in the pre-allocated memory
 * Uses that convention that GrayPair ( i=0, j=0, frequency=0) means
 * available memory
 */
__device__ inline void GLCM::insertElement(GrayPair* grayPairs, const GrayPair actualPair, uint& lastInsertionPosition){
    int position = 0;
    // Find if the element was already inserted, and where
    while((!grayPairs[position].compareTo(actualPair)) && (position < numberOfPairs))
        position++;
    // If found
    if((lastInsertionPosition > 0) // 0,0 as first element will increase insertion position
        && (position != numberOfPairs)){ // if the item was already inserted
        grayPairs[position].operator++();
        if((actualPair.getGrayLevelI() == 0) && (actualPair.getGrayLevelJ() == 0)
            && (grayPairs[position].getFrequency() == actualPair.getFrequency()))
            // corner case, inserted pair 0,0 that matches with every empty field
            lastInsertionPosition++;
    }
    else
    {
        grayPairs[lastInsertionPosition] = actualPair;
        lastInsertionPosition++;
    }
}

/*
    This method creates array of GrayPairs
*/
__device__ void GLCM::initializeGlcmElements() {
    // Define subBorders offset depending on orientation
    int initialWindowColumnOffset = computeWindowColumnOffset();
    int initialWindowRowOffset = computeWindowRowOffset();

    grayLevelType referenceGrayLevel;
    grayLevelType neighborGrayLevel;
    unsigned int lastInsertionPosition = 0;
    // Navigate the sub-window of interest
    for (int i = 0; i < getWindowRowsBorder() ; i++)
    {
        for (int j = 0; j < getWindowColsBorder(); j++)
        {
            // Extract the two pixels in the pair
            int referenceIndex = getReferenceIndex(i, j,
                    initialWindowRowOffset, initialWindowColumnOffset);
            // Application limit: only up to 2^16 gray levels
            referenceGrayLevel = pixels[referenceIndex]; // should be safe
            int neighborIndex = getNeighborIndex(i, j,
                    initialWindowColumnOffset);
            // Application limit: only up to 2^16 gray levels
            neighborGrayLevel = pixels[neighborIndex];  // should be safe

            GrayPair actualPair(referenceGrayLevel, neighborGrayLevel);
            insertElement(grayPairs, actualPair, lastInsertionPosition);

            if(windowData.symmetric) // Create the symmetric counterpart
            {
                GrayPair simmetricPair(neighborGrayLevel, referenceGrayLevel);
                insertElement(grayPairs, simmetricPair, lastInsertionPosition);
            }
            
        }
    }
    effectiveNumberOfGrayPairs = lastInsertionPosition;
    codifyAggregatedPairs();
    codifyMarginalPairs();
}

/* Method that inserts a AggregatedGrayPair in the pre-allocated memory
 * Uses that convention that AggregateGrayPair (k=0, frequency=0) means
 * available memory
 */
__device__ inline void GLCM::insertElement(AggregatedGrayPair* elements, const AggregatedGrayPair actualPair, uint& lastInsertionPosition){
    int position = 0;
    // Find if the element was already inserted, and where
    while((!elements[position].compareTo(actualPair)) && (position < numberOfPairs))
        position++;
    // If found
    if((lastInsertionPosition > 0) && // corner case 0 as first element
        (position != numberOfPairs)){ // if the item was already inserted
            elements[position].increaseFrequency(actualPair.getFrequency());
        if((actualPair.getAggregatedGrayLevel() == 0) && // corner case 0 as regular element
        (elements[position].getFrequency() == actualPair.getFrequency()))
            // corner case, inserted 0 that matches with every empty field
            lastInsertionPosition++;
    }
    else
    {
        elements[lastInsertionPosition] = actualPair;
        lastInsertionPosition++;
    }
}

/* This method will produce the 2 arrays of AggregatedPairs (k, frequency)
 * where k is the sum or difference of both grayLevels of 1 GrayPair.
 * This representation is used in computeSumXXX() and computeDiffXXX() features
*/
__device__ void GLCM::codifyAggregatedPairs() {
    unsigned int lastInsertPosition = 0;
    // summed pairs first
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        // Create summed pairs first
        grayLevelType k= grayPairs[i].getGrayLevelI() + grayPairs[i].getGrayLevelJ();
        AggregatedGrayPair summedElement(k, grayPairs[i].getFrequency());

        insertElement(summedPairs, summedElement, lastInsertPosition);
    }
    numberOfSummedPairs = lastInsertPosition;

    // diff pairs
    lastInsertPosition = 0;
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        int diff = grayPairs[i].getGrayLevelI() - grayPairs[i].getGrayLevelJ();
        grayLevelType k= static_cast<uint>(abs(diff));
        AggregatedGrayPair element(k, grayPairs[i].getFrequency());

        insertElement(subtractedPairs, element, lastInsertPosition);
    }
    numberOfSubtractedPairs = lastInsertPosition;
}

/* This method will produce the 2 arrays of AggregatedPairs (k, frequency)
 * where k is one grayLevel of GLCM and frequency is the "marginal" frequency of that level
 * (ie. how many times k is present in all GrayPair<k, ?>)
 * This representation is used for computing features HX, HXY, HXY1, imoc
*/
__device__ void GLCM::codifyMarginalPairs() {
    unsigned int lastInsertPosition = 0;
    // xMarginalPairs first
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        grayLevelType firstGrayLevel = grayPairs[i].getGrayLevelI();
        AggregatedGrayPair element(firstGrayLevel, grayPairs[i].getFrequency());

        insertElement(xMarginalPairs, element, lastInsertPosition);
    }
    numberOfxMarginalPairs = lastInsertPosition;

    // yMarginalPairs second
    lastInsertPosition = 0;
    for(int i = 0 ; i < effectiveNumberOfGrayPairs; i++){
        grayLevelType secondGrayLevel = grayPairs[i].getGrayLevelJ();
        AggregatedGrayPair element(secondGrayLevel, grayPairs[i].getFrequency());

        insertElement(yMarginalPairs, element, lastInsertPosition);
    }
    numberOfyMarginalPairs = lastInsertPosition;
}

/* DEBUGGING METHODS */
__device__ void GLCM::printGLCM() const {
    printGLCMData();
    printGLCMElements();
    printAggregated();
    printMarginalProbabilityElements();
}

__device__ void GLCM::printGLCMData() const{
    printf("\n");
    printf("***\tGLCM Data\t***\n");
    printf("Shift rows: %d \n", windowData.shiftRows);
    printf("Shift columns: %d \n", windowData.shiftColumns);
    printf("Father Window side: %d \n", windowData.side);
    printf("Border Rows: %d \n", getWindowRowsBorder());
    printf("Border Columns: %d \n", getWindowColsBorder());
    printf("Simmetric: ");
    if(windowData.symmetric){
    	printf("Yes\n");
    }
    else{
    	printf("No\n");
    }
    printf("\n");;
}

__device__ void GLCM::printGLCMElements() const{
    printf("* GrayPairs *\n");
    for (int i = 0; i < effectiveNumberOfGrayPairs; ++i) {
        grayPairs[i].printPair();;
    }
}

__device__ void GLCM::printAggregated() const{
    printGLCMAggregatedElements(true);
    printGLCMAggregatedElements(false);
}

__device__ void GLCM::printGLCMAggregatedElements(bool areSummed) const{
    printf("\n");
    if(areSummed) {
        printf("* Summed grayPairsMap *\n");
        for (int i = 0; i < numberOfSummedPairs; ++i) {
            summedPairs[i].printPair();
        }
    }
    else {
        printf("* Subtracted grayPairsMap *\n");
        for (int i = 0; i < numberOfSubtractedPairs; ++i) {
            subtractedPairs[i].printPair();
        }
    }
}



__device__ void GLCM::printMarginalProbabilityElements() const{
    printf("\n* xMarginal Codifica\n");
    for (int i = 0; i < numberOfxMarginalPairs; ++i) {
        printf("(%d, X):\t%d\n", xMarginalPairs[i].getAggregatedGrayLevel(), xMarginalPairs[i].getFrequency());
    }
    printf("\n* yMarginal Codifica\n");
    for (int i = 0; i <numberOfyMarginalPairs; ++i) {
        printf("(X, %d):\t%d\n", yMarginalPairs[i].getAggregatedGrayLevel(), yMarginalPairs[i].getFrequency());

    }

}


